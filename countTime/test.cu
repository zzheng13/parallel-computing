#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : prog4.c:  Program Assignment 4 Serial Code
 Version     :
 Copyright   :
 Description : Gaussian Elimiation Code to solve a system of linear equations
 ============================================================================
 */
#include<time.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#define N 500
void forwardElimination(double mat[N][N+1]);
int forwardElim(double mat[N][N+1]);

//Prints a matrix
void printMat(double mat[N][N+1]);


#include "./inputArray500.txt"


/*-------------------------------------------------------------------
 * Function:    main
 * Purpose:     initializes matrix and vector for gaussian elimination
 * In args:     none
 * Outputs:     None
 *
 */
int main()
{
    clock_t begin =clock();
   // double x[N];  // An array to store solution

    printf("print mat:\n");
    printMat(mat);

    //use forward elimination to create diagonal matrix
    forwardElimination(mat);
    
    double* d_A, * d_B;
    int jj,i, j;

    double A[N][N];
    double B[N];

    //this is where you assign data to the matrixes

    //for loop with a counter
    //start with matrix B and use mat 

    //for loop with another counter
    //start with matrix A and use mat 


    for (i=0; i < N;i++){
        B[i]=mat[i][N];
        for (j=0;j<N;j++){
            A[i][j]=mat[i][j];
        }
    }

  

    
  
    hipMalloc((void **) &d_A, N*N*sizeof(double));
    hipMemcpy(d_A, A, N*N*sizeof(double), hipMemcpyHostToDevice);
    hipMalloc((void **) &d_B, N*sizeof(double));
    hipMemcpy(d_B, B, N*sizeof(double), hipMemcpyHostToDevice);

    hipblasDtrsm('l','l','t','n', N, N, 1.0, d_A, N, d_B, N);

    for(jj=0; jj < 4; jj++) B[jj]=0.0;


     


    hipDeviceSynchronize();
    
    hipMemcpy(B, d_B, N*sizeof(double), hipMemcpyDeviceToHost);
    
    hipFree(d_A);
    
    hipFree(d_B);
    
    printf("\nSolution to Linear Equations\n");
    for (int row = 0; row < N; row++)
        printf("x[%d] = %f\n", row, B[row]);




    //from textbook programming problem 5.4 method 1
    //Row oriented back substitution to find solution to linear equations
    /*
    for(int row = n-1; row >= 0; row--) {
        x[row] = b[row];
        for(int col = row+1; col < n; col++)
            x[row] -= mat[row][col] * x[col];
        x[row] /= mat[row][row];
    }
    */

    //from textbook programming problem 5.4 method 2
    //Column oriented back substitutionto find solution to linear equations
    
    //printf("\nSolution to Linear Equations\n");
    //for (int row = 0; row < N; row++)
        //printf("x[%d] = %f\n", row, B[row]);
    clock_t end = clock();
    double time_spent = (double) (end-begin)/CLOCKS_PER_SEC;
    printf("The time spent is :%lf\n", time_spent);
    return 0;
}


/*-------------------------------------------------------------------
 * Function:    forwardElimination
 * Purpose:     calculates triangular matrix
 * In args:     matrix derived from system of linear equations
 * Outputs:     triangularized matrix
 *
 */
// function to get matrix content
void forwardElimination(double mat[N][N+1])
{
    /* reduction into r.e.f. */
    int singular_flag = forwardElim(mat);

    /* if matrix is singular */
    if (singular_flag != -1)
    {
        printf("Singular Matrix.\n");

        /* if the RHS of equation corresponding to
           zero row  is 0, * system has infinitely
           many solutions, else inconsistent*/
        if (mat[singular_flag][N])
            printf("Inconsistent System.");
        else
            printf("May have infinitely many "
                   "solutions.");

        exit(1);
    }

}
/*-------------------------------------------------------------------
 * Function:    swap_row
 * Purpose:     function for elemntary operation of swapping two rows
 * In args:     matrix derived from system of linear equations, row numbers i and j
 * Outputs:     matrix with swapped rows
 *
 */

// function for elemntary operation of swapping two rows
void swap_row(double mat[N][N+1], int i, int j)
{
    //printf("Swapped rows %d and %d\n", i, j);

    for (int k=0; k<=N; k++)
    {
        double temp = mat[i][k];
        mat[i][k] = mat[j][k];
        mat[j][k] = temp;
    }
}


/*-------------------------------------------------------------------
 * Function:    printMat
 * Purpose:     function to print matrix content at any stage
 * In args:     matrix derived from system of linear equations
 * Outputs:     none
 *
 */
void printMat(double mat[N][N+1])
{
    for (int i=0; i<N; i++, printf("\n"))
        for (int j=0; j<=N; j++)
            printf("%.1lf ", mat[i][j]);

    printf("\n");
}

/*-------------------------------------------------------------------
 * Function:    forwardElim
 * Purpose:     function to reduce matrix to r.e.f. triangular form
 * In args:     matrix derived from system of linear equations
 * Outputs:     triangularized matrix
 *
 *///
int forwardElim(double mat[N][N+1])
{
    for (int k=0; k<N; k++)
    {
        // Initialize maximum value and index for pivot
        int i_max = k;
        int v_max = mat[i_max][k];

        /* find greater amplitude for pivot if any */
        for (int i = k+1; i < N; i++)
            if (abs(mat[i][k]) > v_max)
                v_max = mat[i][k], i_max = i;

        /* if a prinicipal diagonal element  is zero,
         * it denotes that matrix is singular, and
         * will lead to a division-by-zero later. */
        if (!mat[k][i_max])
            return k; // Matrix is singular

        /* Swap the greatest value row with current row */
        if (i_max != k)
            swap_row(mat, k, i_max);


        for (int i=k+1; i<N; i++)
        {
            /* factor f to set current row kth elemnt to 0,
             * and subsequently remaining kth column to 0 */
            double f = mat[i][k]/mat[k][k];

            /* subtract fth multiple of corresponding kth
               row element*/
            for (int j=k+1; j<=N; j++)
                mat[i][j] -= mat[k][j]*f;

            /* filling lower triangular matrix with zeros*/
            mat[i][k] = 0;
        }

        //print(mat);        //for matrix state
    }
    //print(mat);            //for matrix state
    return -1;
}

